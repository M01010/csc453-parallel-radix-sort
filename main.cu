#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 3
#define MAX_DIGITS 3


__device__ int digit_of(int number, int digit) {
    return number / (int)pow(10, digit - 1) % 10;
}


__global__ void radix_sort(int *idata, int *odata, int size, int d) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int index_number = digit_of(idata[index], d);

    int before = 0;
    int after = 0;

    int cur_idx = 0;

    while (cur_idx < index) {
        int cur_number = digit_of(idata[cur_idx], d);
        if (cur_number <= index_number)
            before++;
        cur_idx++;
    }
    while (cur_idx < size) {
        int cur_number = digit_of(idata[cur_idx], d);
        if (cur_number < index_number)
            after++;
        cur_idx++;
    }

    odata[before+after] = idata[index];
}


void random_ints(int *data, int size) {
    for (int i = 0; i < size; i++) {
        double random = (double) rand() / RAND_MAX; // Random value in [0, 1]
        int num = (int) (random * pow(10, MAX_DIGITS));
        data[i] = num;
    }
}


void show_array(int *data, int size) {
    printf(" {");
    for (int i = 0; i < size - 1; i++) {
        printf("%d, ", data[i]);
    }
    printf("%d}\n", data[size - 1]);
}


int main() {
    srand(time(NULL));

    int *h_idata, *h_odata;
    int *d_idata, *d_odata;
    size_t size = N * N * sizeof(int);

    h_idata = (int *)malloc(size);
    h_odata = (int *)malloc(size);

    random_ints(h_idata, N * N);

    hipMalloc((void **)&d_idata, size);
    hipMalloc((void **)&d_odata, size);

    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);

    printf("Initial Array:");
    show_array(h_idata, N*N);

    for (int d = 1; d <= MAX_DIGITS; d++) {
        radix_sort<<<N, N>>>(d_idata, d_odata, N*N, d);
        hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);

        printf("Step %d:", d);
        show_array(h_odata, N*N);

        int *temp = d_idata;
        d_idata = d_odata;
        d_odata = temp;
        hipDeviceSynchronize();
    }

    if (MAX_DIGITS % 2 == 0)
        hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);
    else
        hipMemcpy(h_odata, d_idata, size, hipMemcpyDeviceToHost);

    printf("Sorted Array:");
    show_array(h_odata, N*N);

    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);
    return 0;
}